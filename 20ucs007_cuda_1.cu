#include "hip/hip_runtime.h"
//20ucs007
//Abhishek Suman

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

static const int Size = 2;

__global__ void add(const int *arr, int *out, const int size) {
    int indie = threadIdx.x;
    int sum = 0;
    for (int i = indie; i < size; i += Size)
        sum += arr[i];

    __shared__ int t[2];
    if(indie<2){
        t[indie]=sum;
    }
    if(indie==0)  {*out = t[0]; *(out+1) = t[1];}
}

void cudaAdd(int* b, int* a, int size) {
    int* ptr_a = NULL;
    int* ptr_b = NULL;

    hipMalloc((void**)&ptr_b, size * sizeof(int));
    hipMalloc((void**)&ptr_a, size * sizeof(int));

    hipMemcpy(ptr_a, a, size * sizeof(int), hipMemcpyHostToDevice);


      add<<<2, (size+1)/2>>>(ptr_a, ptr_b, size);

    hipMemcpy(b, ptr_b, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(ptr_b);
    hipFree(ptr_a);
}

int main(int argc, char** argv) {
    int count;
    printf("\n");
    printf("Input array size : ");
    scanf("%d", &count);
    if(count<1){
    	printf("Not a valid number. \n");
    	return 0;
    }
    int a[count];
    srand(time(NULL));
    for(int i=0;i<count;i++){
      a[i]=rand()%1000;
    }
    int b[count] = {0};

    cudaAdd(b, a, count);
    printf("\n Random generated array: \n");
      for(int i=0;i<count;i++){
      printf("%d ", a[i]);
    }
      printf("\n\nAddition of even indices: %d\n", b[0]);
      printf("Addition of odd indices : %d\n", b[1]);
      printf("\n");

    return 0;
}
